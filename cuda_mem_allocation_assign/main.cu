#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <cstring>

__global__ void mem_trans_test_up(int * input, int array_size){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < array_size)
        printf("tid : %d, gid : %d, value : %d \n",threadIdx.x, gid, input[gid]);
}

int main()
{
    int size = 64;
	int byte_size = sizeof(int) * size;

	int * h_input;
    h_input = (int*)malloc(byte_size);       //allocating memory for host          //malloc returns void pointer by default so we have to cast it to int.

    time_t t;
    srand((unsigned)time(&t));
    for(int i = 0; i < size; i++){
        h_input[i] = (int)(rand() & 0xff);
    }

    int * d_input;
    //allocating memory in device        // now we will use CUDA memory functions

	hipMalloc((void**)&d_input, byte_size);
    // transfer the initiallized array in host to device using cudaMemcpy
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);
    //kernal parameters
    dim3 block(2);
    dim3 grid(5);
    //kernal launch
    mem_trans_test_up <<<grid, block>>>(d_input,size);
    //wait until kernal processing finished
    hipDeviceSynchronize();
    //reclaiming memory
    hipFree(d_input);
    free(h_input);
}
