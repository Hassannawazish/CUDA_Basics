#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void cudd(){                                                       // it is a kernal/device code
	printf("Hasan Nawazish printing using cuda \n");
}

__global__ void print_blockids(){
	printf("blockIDx.x : %d, blockIDx.y : %d, blockIDx.z : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDim.y : %d \n",
		blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

__global__ void print_threadids(){
	printf("threadIDx.x : %d, threadIDx.y : %d, threadIDx.z : %d \n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void unique_idx_calc_threadIdx(int * input){
	int tid = threadIdx.x;
	printf("threadIdx : %d, value : %d \n", tid, input[tid]);
}

int main(){                                                                   // Host Code
	// int nx(16), ny(16);

	// dim3 block(8,8);
	// dim3 grid(nx / block.x, ny / block.y);
	// print_blockids <<<grid,block>>>();

    int array_size = 8;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,1,3,56,43,76,76,8};
	for(int i=0; i< array_size; i++){
		printf("%d", h_data[i]);
	}
	printf("\n \n");

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
	dim3 block(4);
	dim3 grid(2);

	unique_idx_calc_threadIdx <<< grid, block >>> (d_data);


	hipDeviceSynchronize();                                                      // wait untile previously launched kernals finishes
	hipDeviceReset();
}
 