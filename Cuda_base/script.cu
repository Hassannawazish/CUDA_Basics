#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void cudd(){                                                       // it is a kernal/device code
	printf("Hasan Nawazish printing using cuda");
}

int main(){                                                                   // Host Code
cudd <<<1,1>>>();
hipDeviceSynchronize();                                                      // wait untile previously launched kernals finishes
hipDeviceReset();
}
