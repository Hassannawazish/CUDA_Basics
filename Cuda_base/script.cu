#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void cudd(){                                                       // it is a kernal/device code
	printf("Hasan Nawazish printing using cuda \n");
}

__global__ void print_threadids(){
	printf("threadIDx.x : %d, threadIDx.y : %d, threadIDx.z : %d \n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(){                                                                   // Host Code
	int nx(16), ny(16);

	dim3 block(8,8);
	dim3 grid(nx / block.x, ny / block.y);
	print_threadids <<<grid,block>>>();

	hipDeviceSynchronize();                                                      // wait untile previously launched kernals finishes
	hipDeviceReset();
}
 
